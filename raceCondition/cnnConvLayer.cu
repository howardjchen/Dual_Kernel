#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
#include <iostream>
#include "cnnConvLayer.h"
#include <stdio.h>
#include <unistd.h>
using namespace std;

#define xDim 512
#define yDim 32
#define zDim 32

#define xThreadDim 16
#define yThreadDim 16
#define zThreadDim 4


int outputsize = 512*16*16;
int Outputsize = xDim*yDim*zDim;

int *devoutNeu;
int *devPooling;
short *devFilt;
short *devinNeu;
int *outResult = new int[outputsize]();
int *outResult_neu = new int[Outputsize]();

// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int sum, ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int fmArea = FMSIZE *FMSIZE;
	int outArea = FMSIZE/2 * FMSIZE/2;


	cout << "convolutioning..." << endl;

	// Convolution
	for(fn = 0; fn < FILTNUM; fn++) //512
	{
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE) //32
		{
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE)  //32
			{
				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++)  //512
				{
					for(y = 0; y < FILTSIZE; y++)  //3
					{
						for(x = 0; x < FILTSIZE; x++)  //3
						{
							ifmy = fmy - FILTSIZE / 2 + y;		//no dependancy
							ifmx = fmx - FILTSIZE / 2 + x;		//no dependancy
							filtIdx = (fn * filtVol) + (sli * filtArea) + (y * FILTSIZE) + x;	//no dependancy
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;							//no dependancy
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)		
								sum += filt[filtIdx] * inNeu[inNeuIdx];
						}
					}
				}
				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}


 	cout << "Pooling....." << endl;
	// Max Pooling with Window Size 2x2
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++)
	{
		for(fmy = 0; fmy < FMSIZE/2 ; fmy += 1)
		{
			for(fmx = 0; fmx < FMSIZE/2 ; fmx += 1)
			{
				outNeuIdx = sli*fmArea + fmy*2*FMSIZE + fmx*2;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 2; y++)
				{
					for(x = 0; x < 2; x++)
					{
						ofmy = fmy*2 + y;
						ofmx = fmx*2 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/2 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}


void initGPU()
{
	int outNeuVol = FILTNUM * FMSIZE * FMSIZE;  //512x32x32
	int outPolVol = FILTNUM * FMSIZE/2 * FMSIZE/2;  //512x16x16
	int filtTensorVol = sizeof(short)*FILTNUM*FMDEPTH*FILTSIZE*FILTSIZE; //512x512x3x3
	int inNeuVol = sizeof(short)*FMDEPTH*FMSIZE*FMSIZE;	//512x32x32

	hipMalloc(&devoutNeu, sizeof(int)*outNeuVol);
	hipMalloc(&devPooling, sizeof(int)*outPolVol);
	hipMalloc(&devFilt, filtTensorVol);
	hipMalloc(&devinNeu, inNeuVol);

	hipMemcpy(devFilt, filt, filtTensorVol, hipMemcpyHostToDevice);
	hipMemcpy(devinNeu, inNeu, inNeuVol, hipMemcpyHostToDevice);
}


/***	Implement your CUDA Kernel here	***/
__global__
void convLayerGPU(short *FILT, short *InNeu, int *outNeural, int *outPooling)
{
	int threadX = threadIdx.x + blockIdx.x * blockDim.x;
	int threadY = threadIdx.y + blockIdx.y * blockDim.y;
	int threadZ = threadIdx.z + blockIdx.z * blockDim.z;
	//int xall = blockDim.x * gridDim.x;
	//int yall = blockDim.y * gridDim.y;
	//int GlobalThreadId = threadX + threadY * xall + threadZ * xall * yall;
	//int GlobalBlockId = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x;

	int sli,y, x;
	int ifmy, ifmx;
	int filtIdx, inNeuIdx, outNeuIdx;
	int filtVol = 4608;  	//512x3x3
	int filtArea = 9;		//3x3
	int fmArea = 1024;	//32x32
	int outArea = 256;	//32/2*32/2
	int sum = 0;

	for(sli = 0; sli < 512; sli++)  //512
	{
		for(y = 0; y < 3; y++)  //3
		{
			for(x = 0; x < 3; x++)  //3
			{
				ifmy = threadY - 3 / 2 + y;		//no dependancy
				ifmx = threadZ - 3 / 2 + x;		//no dependancy
				filtIdx = (threadX * filtVol) + (sli * filtArea) + (y * 3) + x;//no dependancy
				inNeuIdx = sli * fmArea + ifmy * 32 + ifmx;					//no dependancy
				if(ifmy >= 0 && ifmy < 32 && ifmx >= 0 && ifmx < 32)		
					sum += FILT[filtIdx] * InNeu[inNeuIdx];
			}
		}
	}

	// Activation - ReLU
	outNeuIdx = threadX * fmArea + threadY*32 + threadZ;

	if(sum <= 0)
		outNeural[outNeuIdx] = 0;
	else
		outNeural[outNeuIdx] = sum;



	__syncthreads();

 /*========== Max Pooling with Window Size 2x2 =================*/
	
	if(threadX == 0 && threadY == 0 && threadZ == 0 )  //asking 1 thread to do pooling
	{		
		int max, tmpVal, py, px;
		int  ofmy, ofmx, outIdx; // pooling varable
		int xx,yy,slii;

		for(slii = 0; slii < 512; slii++)	//FILTNUM
		{
			for(py = 0; py < 16 ; py += 1) //FMSIZE/2
			{
				for(px = 0; px < 16 ; px += 1)  //FMSIZE/2
				{
					outNeuIdx = slii*fmArea + py*2*32 + px*2;
					max = outNeural[outNeuIdx];
					for(yy = 0; yy < 2; yy++)
					{
						for(xx = 0; xx < 2; xx++)
						{
							ofmy = py*2 + yy;
							ofmx = px*2 + xx;
							outNeuIdx = slii*fmArea + ofmy*32 + ofmx;
							tmpVal = outNeural[outNeuIdx];	
							if(tmpVal > max)
								max = tmpVal;
						}
					}
					outIdx = slii*outArea + py*32/2 + px;
					outPooling[outIdx] = max;
				}
			}
		}
	}
}




/*
__global__ 
void MaxPoolingGPU(int *out)  // Max Pooling with Window Size 2x2
{
	int threadX = threadIdx.x + blockIdx.x * blockDim.x;
	int threadY = threadIdx.y + blockIdx.y * blockDim.y;
	int threadZ = threadIdx.z + blockIdx.z * blockDim.z;
	int xall = blockDim.x * gridDim.x;
	int yall = blockDim.y * gridDim.y;
	int GlobalThreadId = threadX + threadY * xall + threadZ * xall * yall;
	int GlobalBlockId = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x;
	
	int max, tmpVal, outNeuIdx, x, y;
	int fmArea = 32 *32;
	int outArea = 32/2 * 32/2;
	int  ofmy, ofmx, outIdx; // pooling varable

	outNeuIdx = threadX*fmArea + threadY*2*32 + threadZ*2;
	max = outNeu[outNeuIdx];
	for(y = 0; y < 2; y++)
	{
		for(x = 0; x < 2; x++)
		{
			ofmy = threadY*2 + y;
			ofmx = threadZ*2 + x;
			outNeuIdx = threadX*fmArea + ofmy*32 + ofmx;
			tmpVal = outNeu[outNeuIdx];	
			if(tmpVal > max)
				max = tmpVal;
		}
	}
	outIdx = threadX*outArea + threadY*32/2 + threadZ;
	out[outIdx] = max;
}
*/

int main()
{
	float convLayerCPUExecTime, convLayerGPUExecTime;
	init();
		


	timespec time_begin, time_end;                                                 
  	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerCPU();
  	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << " ================ Result ===================" << endl;
	cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;



 	initGPU();
 	dim3 threadPerBlock(xThreadDim, yThreadDim, zThreadDim);
 	dim3 numBlocks(xDim/xThreadDim, yDim/yThreadDim, zDim/zThreadDim);
 	clock_gettime(CLOCK_REALTIME, &time_begin);


	convLayerGPU<<<numBlocks,threadPerBlock>>>(devFilt, devinNeu, devoutNeu, devPooling); 


	hipDeviceSynchronize(); 
  	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = " << convLayerGPUExecTime / 1000 << "ms" << endl;


	int outSize = sizeof(int)*outputsize;
	hipMemcpy(outGPU, devPooling, outSize, hipMemcpyDeviceToHost);
	
	//int OutSize = sizeof(int)*Outputsize; 
	//hipMemcpy(outResult_neu, devoutNeu, OutSize, hipMemcpyDeviceToHost);


	// check the Output of Neu 
	/*for (int i = 0; i < 512*32*32; ++i)
	{
		if (outNeu[i] == outResult_neu[i])
		{
			printf("wrong at =  %d \n", i);
			break;
		}
	}
	printf("PASS!!!\n");*/
	// check the Output of GPU 
	/*for (int i = 0; i < 512*16*16; ++i)
	{
		if (outCPU[i] != outGPU[i])
		{
			printf("wrong at =  %d \n", i);
			break;
		}
	}
	printf("PASS!!!\n");*/



	if(checker())
	{
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	hipFree(&devoutNeu);
	hipFree(&devPooling);
	hipFree(&devFilt);
	hipFree(&devinNeu);

	delete [] outResult;
	delete [] outResult_neu;
	ending();
	
	return 0;
}
